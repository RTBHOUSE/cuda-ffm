#include "hip/hip_runtime.h"
#include "constants.h"
#include "ffm_trainer.h"
#include "cuda_utils.h"

__constant__ float cLearningRate[1];
__constant__ float cL2Reg[1];
__constant__ float cNormalizationFactor[1];       // 1.0 / numFields;
__constant__ float cScaledNormalizationFactor[2]; // samplingFactor * normalizationFactor, idx 0 for y = -1, idx 1 for y = 1
__constant__ int cRowSize[1];                     // numFields * FactorSize

// Fills matrix with constant value
__global__ void fillKernel(float *__restrict__ matrix, const float value)
{
    matrix[blockIdx.x * blockDim.x + threadIdx.x] = value;
}

// Updates FFM weights
//
// Loss function and gradients:
//
//    y          := 1 or -1
//    t          := sum_j1(sum_j2(w[j1, f2] * w[j2, f1]))
//    p(y=1|x)   := 1 / (1 + exp(-t))
//
//    L          := p(y=1|x)^[y == 1] * (1 - p(y=1|x))^[y == -1] // likelihood
//               := 1 / (1 + exp(-yt))
//
//    LL         := log(L) // log-likelihood
//               := -log(1 + exp(-yt))
//
//    reg        := 1/2 * learningRate * sum_i(w_i^2) // regularization
//
//    loss       := LL + reg
//
//    -grad[w]   := -dLoss / dw == (dLL / dt * dt / dw) + dReg / dw
//    kappa      := -dLL / dt == -y / (1 + exp(yt)) ==  -y * exp(-yt) / (1 + exp(-yt))
//    dt / dw1   := w2
//    dReg / dw1 := learningRate * w1
//
//    -grad[w1]  := kappa * w2 + learningRate * w1
//
// AdaGrad update:
//
//    update[w, i]  := -grad[w] * learningRate / sqrt(sum(j=1..i-1, grad[w, j]^2))
//
__global__ void updateKernel(const float *__restrict__ fieldSums, float *__restrict__ weights, float *__restrict__ squaredGradsSum,
                             const int *__restrict__ input, const float y, int numFields)
{
    __shared__ float _normalizedKappa;

    const int fieldIdx1 = threadIdx.x / 4;
    const int d = threadIdx.x % 4;
    const int fieldIdx2 = blockIdx.x;
    const int rowSize = *cRowSize;

    // sum partially computed t (outer sum)

    typedef hipcub::BlockReduce<float, MaxUpdateBlockSize> BlockReduce;
    __shared__ typename BlockReduce::TempStorage tempStorage;
    const float t = BlockReduce(tempStorage).Sum(fieldSums[threadIdx.x], numFields);

    // compute kappa

    if (threadIdx.x == 0) {
        const float expNegYT = expf(-y * t);
        const float kappa = -y * expNegYT / (1.0f + expNegYT);
        const float normalizationFactor = cScaledNormalizationFactor[y > 0];
        CUDA_ASSERT_FIN(expNegYT);
        CUDA_ASSERT_FIN(kappa);
        _normalizedKappa = kappa * normalizationFactor;
    }

    __syncthreads();

    // update weights

    if (fieldIdx2 > fieldIdx1) {
        const float normalizedKappa = _normalizedKappa;
        const float l2Reg = *cL2Reg;
        const float learningRate = *cLearningRate;

        CUDA_ASSERT_FIN(normalizedKappa);

        const int j1 = input[fieldIdx1];
        const int j2 = input[fieldIdx2];

        const int offset1 = j1 * rowSize + fieldIdx2 * FactorSize;
        const int offset2 = j2 * rowSize + fieldIdx1 * FactorSize;

        float *weight1Start = weights + offset1;
        float *weight2Start = weights + offset2;

        float *squaredGradsSum1Start = squaredGradsSum + offset1;
        float *squaredGradsSum2Start = squaredGradsSum + offset2;

        float weight1 = weight1Start[d];
        float weight2 = weight2Start[d];

        float prevSquaredGrads1Sum = squaredGradsSum1Start[d];
        float prevSquaredGrads2Sum = squaredGradsSum2Start[d];

        float regTerm1 = weight1 * l2Reg;
        float regTerm2 = weight2 * l2Reg;

        const float grad1 = regTerm1 + normalizedKappa * weight2;
        const float grad2 = regTerm2 + normalizedKappa * weight1;

        prevSquaredGrads1Sum += grad1 * grad1;
        prevSquaredGrads2Sum += grad2 * grad2;

        CUDA_ASSERT_FIN(prevSquaredGrads1Sum);
        CUDA_ASSERT_FIN(prevSquaredGrads2Sum);

        weight1 = weight1 - learningRate * grad1 * rsqrtf(prevSquaredGrads1Sum);
        weight2 = weight2 - learningRate * grad2 * rsqrtf(prevSquaredGrads2Sum);

        weight1Start[d] = weight1;
        weight2Start[d] = weight2;

        squaredGradsSum1Start[d] = prevSquaredGrads1Sum;
        squaredGradsSum2Start[d] = prevSquaredGrads2Sum;
    }
}

// Partially computes t (each thread block computes the inner sum for one field) - for updates
__global__ void ffmInnerSumKernel(const float *__restrict__ weights, const int *__restrict__ input, float *__restrict__ fieldSums)
{
    const int fieldIdx1 = threadIdx.x;
    const int fieldIdx2 = blockIdx.x;

    CUDA_ASSERT(fieldIdx1 < numFields);
    CUDA_ASSERT(fieldIdx2 < numFields);

    const int rowSize = *cRowSize;

    float sum = 0.0f;

    if (fieldIdx2 > fieldIdx1) {
        const int j1 = input[fieldIdx1];
        const int j2 = input[fieldIdx2];

        const int offset1 = j1 * rowSize + fieldIdx2 * FactorSize;
        const int offset2 = j2 * rowSize + fieldIdx1 * FactorSize;

        const float4 W1 = hipcub::ThreadLoad<hipcub::LOAD_DEFAULT>((float4 *) (weights + offset1));
        const float4 W2 = hipcub::ThreadLoad<hipcub::LOAD_DEFAULT>((float4 *) (weights + offset2));

        sum += W1.x * W2.x;
        sum += W1.y * W2.y;
        sum += W1.z * W2.z;
        sum += W1.w * W2.w;
    }

    typedef hipcub::BlockReduce<float, MaxPredictBlockSize> BlockReduce;
    __shared__ typename BlockReduce::TempStorage tempStorage;
    float aggregate = BlockReduce(tempStorage).Sum(sum);

    if (threadIdx.x == 0) {
        fieldSums[blockIdx.x] = aggregate * *cNormalizationFactor;
    }
}

// Partially computes t (each thread block computes the inner sum for one field) - for prediction, batch mode
__global__ void batchFfmInnerSumKernel(const float *__restrict__ weights, const int *__restrict__ input, float *__restrict__ fieldSums,
                                       const int numFields)
{
    const int fieldIdx1 = threadIdx.x;
    const int fieldIdx2 = blockIdx.x;

    const int batchIdx = blockIdx.y;
    const int batchInputOffset = (numFields + 1) * batchIdx;
    const int rowSize = *cRowSize;

    CUDA_ASSERT(fieldIdx1 < numFields);
    CUDA_ASSERT(fieldIdx2 < numFields);

    float sum = 0.0f;

    if (fieldIdx2 > fieldIdx1) {
        const int j1 = input[batchInputOffset + fieldIdx1];
        const int j2 = input[batchInputOffset + fieldIdx2];

        const int offset1 = j1 * rowSize + fieldIdx2 * FactorSize;
        const int offset2 = j2 * rowSize + fieldIdx1 * FactorSize;

        const float4 W1 = hipcub::ThreadLoad<hipcub::LOAD_DEFAULT>((float4 *) (weights + offset1));
        const float4 W2 = hipcub::ThreadLoad<hipcub::LOAD_DEFAULT>((float4 *) (weights + offset2));

        sum += W1.x * W2.x;
        sum += W1.y * W2.y;
        sum += W1.z * W2.z;
        sum += W1.w * W2.w;
    }

    typedef hipcub::BlockReduce<float, MaxPredictBlockSize> BlockReduce;
    __shared__ typename BlockReduce::TempStorage tempStorage;
    float aggregate = BlockReduce(tempStorage).Sum(sum);

    if (threadIdx.x == 0) {
        fieldSums[batchIdx * numFields + blockIdx.x] = aggregate * *cNormalizationFactor;
    }
}

// Computes outer sum (t) and and applies logit function - for prediction
__global__ void batchSigmoidKernel(const float *__restrict__ fieldSums, float *__restrict__ predictionResults, const int numFields)
{
    CUDA_ASSERT(threadIdx.x < numFields);

    typedef hipcub::BlockReduce<float, MaxPredictBlockSize> BlockReduce;
    __shared__ typename BlockReduce::TempStorage tempStorage;
    float t = BlockReduce(tempStorage).Sum(fieldSums[blockIdx.x * numFields + threadIdx.x]);

    if (threadIdx.x == 0) {
        const float p = 1.0f / (1.0f + expf(-t));
        CUDA_ASSERT_FIN(p);
        CUDA_ASSERT(p <= 1.0);
        CUDA_ASSERT(p >= 0.0);
        predictionResults[blockIdx.x] = p;
    }
}

FFMTrainer::FFMTrainer(Model const & model, float samplingFactor, int maxBatchSize, float l2Reg, float learningRate)
        : numFields(model.numFields),
          samplingFactor(samplingFactor),
          maxBatchSize(maxBatchSize),
          weightsSize(HashSpaceSize * model.numFields * FactorSize)
{
    const int MaxNumFields = 1024;
    dWeights = cuda_utils.malloc<float>(weightsSize);
    dSquaredGradsSum = cuda_utils.malloc<float>(weightsSize);

    dXYLearnInputBuffer = cuda_utils.malloc<int>(maxBatchSize * (numFields + 1));
    dLearnFieldSums = cuda_utils.malloc<float>(MaxNumFields);

    dXYPredictInputBuffer = cuda_utils.malloc<int>(maxBatchSize * (numFields + 1));
    dPredictFieldSums = cuda_utils.malloc<float>(maxBatchSize * numFields);
    dPredictResultsBuffer = cuda_utils.malloc<float>(maxBatchSize * (numFields + 1));

    CHECK_ERR(hipGetLastError());
    CHECK_ERR(hipDeviceSynchronize());

    cuda_utils.memcpy(dWeights, model.weights.data(), weightsSize);

    cuda_utils.memcpyToSymbol(cL2Reg, l2Reg);
    cuda_utils.memcpyToSymbol(cLearningRate, learningRate);
    cuda_utils.memcpyToSymbol(cNormalizationFactor, model.normalizationFactor);
    cuda_utils.memcpyToSymbol(cRowSize, numFields * FactorSize);
    cuda_utils.memcpyToSymbol2(cScaledNormalizationFactor, model.normalizationFactor * samplingFactor, model.normalizationFactor);

    cuda_utils.memset(dLearnFieldSums, 0, MaxNumFields);
    cuda_utils.memset(dPredictFieldSums, 0, maxBatchSize * numFields);
    fillKernel<<<HashSpaceSize, numFields * FactorSize>>>(dSquaredGradsSum.get(), 1.0);
}

FFMTrainer::~FFMTrainer()
{
}

void FFMTrainer::learn(int const *hXYBatchBuffer, int batchSize)
{
    assert(batchSize <= maxBatchSize);

    cuda_utils.memcpy(dXYLearnInputBuffer, hXYBatchBuffer, batchSize * (numFields + 1));
    const int row_len = numFields + 1;

    for (int i = 0; i < batchSize; ++i) {
        int y = hXYBatchBuffer[i * row_len + numFields];
        const int *xy = dXYLearnInputBuffer.get() + i * row_len;
        ffmInnerSumKernel<<<numFields, numFields>>>(dWeights.get(), xy, dLearnFieldSums.get());
        updateKernel<<<numFields, numFields * 4>>>(dLearnFieldSums.get(), dWeights.get(), dSquaredGradsSum.get(), xy, y, numFields);

        CHECK_ERR(hipGetLastError());
    }
    CHECK_ERR(hipDeviceSynchronize());
}

void FFMTrainer::predict(int const * hXYBatchBuffer, int batchSize, float * predictResults)
{
    assert(batchSize <= maxBatchSize);

    cuda_utils.memcpy(dXYPredictInputBuffer, hXYBatchBuffer, batchSize * (numFields + 1));

    batchFfmInnerSumKernel<<<dim3(numFields, batchSize, 1), numFields>>>(dWeights.get(), dXYPredictInputBuffer.get(),
                                                                         dPredictFieldSums.get(), numFields);
    batchSigmoidKernel<<<batchSize, numFields>>>(dPredictFieldSums.get(), dPredictResultsBuffer.get(), numFields);

    cuda_utils.memcpy(predictResults, dPredictResultsBuffer, batchSize);
}

template <typename T>
T *FFMTrainer::createHostBuffer(int size)
{
    return cuda_utils.hostMalloc<T>(size);
}

template <typename T>
void FFMTrainer::destroyHostBuffer(T *hBuffer)
{
    cuda_utils.hostFree(hBuffer);
}

void FFMTrainer::copyWeightsToHost(float *hWeights)
{
    cuda_utils.memcpy(hWeights, dWeights, weightsSize);
}

void FFMTrainer::copyGradsToHost(float *hGrads)
{
    cuda_utils.memcpy(hGrads, dSquaredGradsSum, weightsSize);
}

void FFMTrainerStatic::init()
{
    CHECK_ERR(hipSetDeviceFlags(hipDeviceMapHost));
    CHECK_ERR(hipSetDeviceFlags(hipDeviceScheduleSpin));
    //CHECK_ERR(hipSetDeviceFlags(hipDeviceScheduleYield));
    //CHECK_ERR(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
    CHECK_ERR(hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, 0));
    CHECK_ERR(hipDeviceSetLimit(cudaLimitDevRuntimePendingLaunchCount, 0));
    CHECK_ERR(hipDeviceSetLimit(hipLimitStackSize, 0));
    CHECK_ERR(hipDeviceSetLimit(hipLimitPrintfFifoSize, 0));
    CHECK_ERR(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
}

void FFMTrainerStatic::destroy()
{
    CHECK_ERR(hipDeviceReset());
}

template int *FFMTrainer::createHostBuffer<int>(int);

template float *FFMTrainer::createHostBuffer<float>(int);

template void FFMTrainer::destroyHostBuffer<int>(int *);

template void FFMTrainer::destroyHostBuffer<float>(float *);
